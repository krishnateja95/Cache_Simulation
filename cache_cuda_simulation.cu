#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <unistd.h>

/************************************************************************/

__host__ int read_trace_element(FILE *infile, unsigned *access_type, unsigned *addr)
{
  int result;
  char c;

  result = fscanf(inFile, "%u %x%c", access_type, addr, &c);
  while (c != '\n') {
    result = fscanf(inFile, "%c", &c);
    if (result == EOF) 
      break;
  }
  if (result != EOF)
    return(1);
  else
    return(0);
}

/*************************************************************************/

__global__ void init_arrays(int *cache, int *two_cache, unsigned *cache_tag, unsigned *two_cache_tag, int *valid_bit, int *two_valid_bit, int *dirty_bit, int *two_dirty_bit, int *lru_counter, int* h_lru_count)
{
	if(threadIdx < 512)
	{
	 cache[threadIdx]=0;
	 two_cache[threadIdx]=0;
     cache_tag[threadIdx]=0;
	 two_cache_tag[threadIdx]=0;
     valid_bit[threadIdx]=0;
	 two_valid_bit[threadIdx]=0;
     dirty_bit[threadIdx]=0;
	 two_dirty_bit[threadIdx]=0;	

	 if((threadIdx%2)==0)
		lru_counter[threadIdx]=2;
	 else
		lru_counter[threadIdx]=1;	 
 
	 if(threadIdx<256)
		 h_lru_count[threadIdx] = 2;
	}	
}

/*************************************************************************/

__global__ void read_hit(unsigned *two_way_tag, unsigned *two_cache_tag, int *two_way_index, int *two_valid_bit, int *two_cache_read_hit, int *found, int *temp_lru, int *lru_counter, int *temp_index)
{
	if ((two_way_tag == two_cache_tag[(two_way_index << 1) | a]) && two_valid_bit[(two_way_index << 1) | a]) // the address tag matches cache tag and data is valid, i.e it is what we want
				{			
					two_cache_read_hit++;// then its a hit
					found = 1; // data was found
					// processor uses that data
					temp_lru = lru_counter[(two_way_index << 1) | threadIdx.x]; // keeping track of the lru value of the location which was operated on
					
					temp_index = (two_way_index << 1) | threadIdx.x; // keeping track of the location which was operated on		
				}	
}

__global__ void read_miss_empty(int *two_cache, int *two_way_index, unsigned *two_cache_tag, int *two_valid_bit, int *two_dirty_bit, int *lru_counter, int* h_lru_count, int *temp_lru, int *temp_index, int *data, unsigned *two_way_tag)
{
	if(two_cache[(two_way_index << 1) | threadIdx.x] == 0 && lru_counter[(two_way_index << 1) | threadIdx.x] == h_lru_count[two_way_index]) // place it in empty spot with highest LRU count for empty spot							
						{
							two_cache[(two_way_index << 1) | threadIdx.x] = data; // get the value from memory and store it in location with highest lru value for empty spot
							
							two_cache_tag[(two_way_index << 1) | threadIdx.x] = two_way_tag; // update the tag in cache
							
							two_valid_bit[(two_way_index << 1) | threadIdx.x] = 1; // set the valid bit to 1 as its the first time value is loaded in cache
							
							two_dirty_bit[(two_way_index << 1) | threadIdx.x] = 0; // setting dirty bit to 0 as new value was loaded into cache from memory
							
							h_lru_count[two_way_index]--; // the highest lru count for an empty spot is now decreased by 1
							
							temp_lru = lru_counter[(two_way_index << 1) | threadIdx.x]; // keeping track of the lru value of the location which was operated on
							
							temp_index = (two_way_index << 1) | threadIdx.x; // keeping track of the location which was operated on
							
						}	
	
}

__global__ void read_miss_full(int *two_cache, int *two_way_index, unsigned *two_cache_tag, int *two_valid_bit, int *two_dirty_bit, int *lru_counter, int* h_lru_count, int *temp_lru, int *temp_index, int *data, unsigned *two_way_tag)
{
	if (lru_counter[(two_way_index << 1) | threadIdx.x]==1) // check for location which was least recently used
						{
							
							two_cache[(two_way_index << 1) | threadIdx.x] = data; // replace the location which was least recently used with data from memory
							
							two_cache_tag[(two_way_index << 1) | threadIdx.x] = two_way_tag; // update the tag in cache
							
							temp_lru = lru_counter[(two_way_index << 1) | threadIdx.x]; // keeping track of the lru value of the location which was operated on
							
							temp_index = (two_way_index << 1) | threadIdx.x; // keeping track of the location which was operated on
							
							two_dirty_bit[(two_way_index << 1) | threadIdx.x] = 0; // setting dirty bit to 0 as new value was loaded into cache from memory
							
							two_valid_bit[(two_way_index << 1) | threadIdx.x] = 1; // data in cache is valid
							
						}
	
}

__global__ void write_hit(unsigned *two_way_tag, unsigned *two_cache_tag, int *two_way_index, int *two_valid_bit, int *two_cache_write_hit, int *w_found, int *temp_lru, int *lru_counter, int *temp_index, int *data)
{
	if((two_cache_tag[(two_way_index << 1) | threadIdx.x] == two_way_tag) && (two_valid_bit[(two_way_index << 1) | threadIdx.x] == 1))  // seeing if the location we wanna write to (we know this from tag) is already present 
			  {
				  w_found = 1; // we found it
				  two_cache_write_hit++;
				    
				  two_valid_bit[(two_way_index << 1) | threadIdx.x] = 1; 
				  
				  if(two_cache[(two_way_index << 1) | threadIdx.x] != data) // if the data sent by processor is new, i.e not same as one found for that tag
				  {
					  two_cache[(two_way_index << 1) | threadIdx.x] = data; // write new data in cache
					  two_dirty_bit[(two_way_index << 1) | threadIdx.x] = 1; // set dirty bit to one as new data was found
				  }
				  else
				  { 
					two_dirty_bit[(two_way_index << 1) | threadIdx.x] = 0; // if processor didn't provide new data, set dirty bit to 0
				  }
				  
				temp_lru = lru_counter[(two_way_index << 1) | threadIdx.x]; // keeping track of the lru value of the location which was operated on
				
				temp_index = (two_way_index << 1) | threadIdx.x; // keeping track of the location which was operated on						

				
			  }
	
}

__global__ void write_miss_empty(int *two_cache, int *two_way_index, unsigned *two_cache_tag, int *two_valid_bit, int *two_dirty_bit, int *lru_counter, int* h_lru_count, int *temp_lru, int *temp_index, int *data, unsigned *two_way_tag)
{
	if(two_cache[(two_way_index << 1) | threadIdx.x]==0 && lru_counter[(two_way_index << 1) | threadIdx.x] == h_lru_count[two_way_index]) // we search for an empty location with highest LRU count for an empty spot
					{
						two_cache[(two_way_index << 1) | threadIdx.x] = data; // load the new data given by processor in cache
						
						
						two_cache_tag[(two_way_index << 1) | threadIdx.x] = two_way_tag; // update the tag in cache
						
						two_valid_bit[(two_way_index << 1) | threadIdx.x] = 1; // set the valid bit to 1 as its the first time value is loaded in cache
						
						two_dirty_bit[(two_way_index << 1) | threadIdx.x] = 0; // setting dirty bit to 0 as value loaded in cache was just loaded in memory too
						
						h_lru_count[two_way_index]--; // the highest lru count for an empty spot is now decreased by 1
						
						temp_lru = lru_counter[(two_way_index << 1) | threadIdx.x]; // keeping track of the lru value of the location which was operated on
						
						temp_index = (two_way_index << 1) | threadIdx.x; // keeping track of the location which was operated on
				
					}
	
}

__global__ void write_miss_full(int *two_cache, int *two_way_index, unsigned *two_cache_tag, int *two_valid_bit, int *two_dirty_bit, int *lru_counter, int* h_lru_count, int *temp_lru, int *temp_index, int *data, unsigned *two_way_tag)
{
			if (lru_counter[(two_way_index << 1) | a] == 1) // check for location which was least recently used
							{
								
								two_cache[(two_way_index << 1) | a] = data; // replace the location which was least recently used with data from processor
								
								two_cache_tag[(two_way_index << 1) | a] = two_way_tag; // update the tag in cache
								
								temp_lru = lru_counter[(two_way_index << 1) | a]; // keeping track of the lru value of the location which was operated on
								
								temp_index = (two_way_index << 1) | a; // keeping track of the location which was operated on
								
								two_dirty_bit[(two_way_index << 1) | a] = 0; // setting dirty bit to 0 as the value loaded in cache was also just loaded in memory
								
								two_valid_bit[(two_way_index << 1) | a] = 1; // the data is valid
								
								
							}
	
}

__global__ void lru_count(int *lru_counter, int *two_way_index, int *temp_lru)
{
	if(lru_counter[(two_way_index << 1) | threadIdx.x] > temp_lru) // if any lru value is greater than the value of location operated upon
					lru_counter[(two_way_index << 1) | threadIdx.x]--; // decrement it
	
}

/***************************************************************/

int main(int argc, char** argv)
{

	srand(time(0));
	  
	FILE *trace_file;
	  
	trace_file = fopen(argv[1], "r");
	  
	unsigned address, read;
	  
	int d_mask = 0x1ff;

	int two_mask = 0xff;

	int  cache_read_hit = 0, cache_read_miss = 0,cache_write_miss = 0,cache_write_hit = 0,i,n,j=0; 

	int  two_cache_read_hit = 0, two_cache_read_miss = 0, two_cache_write_miss = 0, two_cache_write_hit = 0;
	
	int *dev_two_cache_read_hit, *dev_two_cache_write_hit;

	int cache[512], two_cache[512];

	int *dev_cache, *dev_two_cache;

	int data;
	
	int *dev_data;

	unsigned two_cache_tag[512],cache_tag[512];

	unsigned *dev_two_cache_tag, *dev_cache_tag;

	int valid_bit[512], dirty_bit[512], two_valid_bit[512], two_dirty_bit[512];

	int *dev_valid_bit, *dev_dirty_bit, *dev_two_valid_bit, *dev_two_dirty_bit;

	unsigned two_way_tag, direct_mapped_tag;

	unsigned *dev_two_way_tag;

	int two_way_index, direct_mapped_index;

	int *dev_two_way_index;

	int lru_counter[512], h_lru_count[256];

	int *dev_lru_counter, *dev_h_lru_count;

	int a;

	int found, w_found;
	
	int *dev_found, *dev_w_found;

	int temp_lru, temp_index;

	int *dev_temp_lru, *dev_temp_index;


	/* for(a=0;a<512;a++)
	   {
		 cache[a]=0;
		 two_cache[a]=0;
		 cache_tag[a]=0;
		 two_cache_tag[a]=0;
		 valid_bit[a]=0;
		 two_valid_bit[a]=0;
		 dirty_bit[a]=0;
		 two_dirty_bit[a]=0;	

		 if((a%2)==0)
			lru_counter[a]=2;
		 else
			lru_counter[a]=1;	 
	 
		 if(a<256)
			 h_lru_count[a] = 2;
	   }
	 */
 
 hipMalloc((void**)&dev_cache, 512*sizeof(int));
 hipMalloc((void**)&dev_two_cache, 512*sizeof(int));
 hipMalloc((void**)&dev_cache_tag, 512*sizeof(unsigned));
 hipMalloc((void**)&dev_two_cache_tag, 512*sizeof(unsigned));
 hipMalloc((void**)&dev_valid_bit, 512*sizeof(int));
 hipMalloc((void**)&dev_two_valid_bit, 512*sizeof(int));
 hipMalloc((void**)&dev_dirty_bit, 512*sizeof(int));
 hipMalloc((void**)&dev_two_dirty_bit, 512*sizeof(int));
 hipMalloc((void**)&dev_lru_counter, 512*sizeof(int));
 hipMalloc((void**)&dev_h_lru_count, 256*sizeof(int));
 
 hipMemcpy(dev_cache, cache, 512*sizeof(int), hipMemcpyHostToDevice);
 hipMemcpy(dev_two_cache, two_cache, 512*sizeof(int), hipMemcpyHostToDevice);
 hipMemcpy(dev_cache_tag, cache_tag, 512*sizeof(unsigned), hipMemcpyHostToDevice);
 hipMemcpy(dev_two_cache_tag, two_cache_tag, 512*sizeof(unsigned), hipMemcpyHostToDevice);
 hipMemcpy(dev_valid_bit, valid_bit, 512*sizeof(int), hipMemcpyHostToDevice);
 hipMemcpy(dev_two_valid_bit, two_valid_bit, 512*sizeof(int), hipMemcpyHostToDevice);
 hipMemcpy(dev_dirty_bit, dirty_bit, 512*sizeof(int), hipMemcpyHostToDevice);
 hipMemcpy(dev_two_dirty_bit, two_dirty_bit, 512*sizeof(int), hipMemcpyHostToDevice);
 hipMemcpy(dev_lru_counter, lru_counter, 512*sizeof(int), hipMemcpyHostToDevice);
 hipMemcpy(dev_h_lru_count, h_lru_count, 256*sizeof(int), hipMemcpyHostToDevice);
 
 init_arrays<<<1,512>>>(dev_cache, dev_two_cache, dev_cache_tag, dev_two_cache_tag, dev_valid_bit, dev_two_valid_bit, dev_dirty_bit, dev_two_dirty_bit, dev_lru_counter, dev_h_lru_count);
 
 hipMemcpy(cache, dev_cache, 512*sizeof(int), hipMemcpyDeviceToHost);
 hipMemcpy(two_cache, dev_two_cache, 512*sizeof(int), hipMemcpyDeviceToHost);
 hipMemcpy(cache_tag, dev_cache_tag, 512*sizeof(unsigned), hipMemcpyDeviceToHost);
 hipMemcpy(two_cache_tag, dev_two_cache_tag, 512*sizeof(unsigned), hipMemcpyDeviceToHost);
 hipMemcpy(valid_bit, dev_valid_bit, 512*sizeof(int), hipMemcpyDeviceToHost);
 hipMemcpy(two_valid_bit, dev_two_valid_bit, 512*sizeof(int), hipMemcpyDeviceToHost);
 hipMemcpy(dirty_bit, dev_dirty_bit, 512*sizeof(int), hipMemcpyDeviceToHost);
 hipMemcpy(two_dirty_bit, dev_two_dirty_bit, 512*sizeof(int), hipMemcpyDeviceToHost);
 hipMemcpy(lru_counter, dev_lru_counter, 512*sizeof(int), hipMemcpyDeviceToHost);
 hipMemcpy(h_lru_count, dev_h_lru_count, 256*sizeof(int), hipMemcpyDeviceToHost);
 
 hipFree(dev_cache);
 hipFree(dev_two_cache);
 hipFree(dev_cache_tag);
 hipFree(dev_two_cache_tag);
 hipFree(dev_valid_bit);
 hipFree(dev_two_valid_bit);
 hipFree(dev_dirty_bit);
 hipFree(dev_two_dirty_bit);
 hipFree(dev_lru_counter);
 hipFree(dev_h_lru_count);
 
while(read_trace_element(trace_file, &read, &address))
	{	 
		if(read == 0 || read == 2)
			read = 1; // reading if read was zero or two as per trace
		else
			read = 0; // writing if read was 1 as per trace

		 data = (rand()%100)+1;

	//Direct Mapped
	
		direct_mapped_index = (address >> 4) & d_mask;
		direct_mapped_tag = address >> 13;
		
		

		if(read)
		{

			if((direct_mapped_tag == cache_tag[direct_mapped_index]) && valid_bit[direct_mapped_index]) // the address tag matches cache tag and data is valid, i.e it is what we want
			{
				
				cache_read_hit++;// then its a hit

			// processor uses that data

			}
			else // if its a miss
			{
		
				 cache_read_miss++;

				 cache[direct_mapped_index] = data; // data loaded from main memory into cache

				 valid_bit[direct_mapped_index] = 1; // now that data is loaded from main memory, valid bit becomes 1 
				 
				 dirty_bit[direct_mapped_index] = 0; // since a fresh data is loaded from main memory, dirty bit is 0;

				 cache_tag[direct_mapped_index] = direct_mapped_tag; // tag is updated in cache with new address' tag

				 // processor uses that data

			}

		}
		else // write statements go over here  
		 //we're implementing write-back and write allocate
		{
			 if(direct_mapped_tag == cache_tag[direct_mapped_index] && valid_bit[direct_mapped_index]) // checking the tag of address to be written to is same as what processor provided 
			   {
					if(cache[direct_mapped_index] != data) // if the data sent by processor is not same as the one already present in that cache location
					{
						 dirty_bit[direct_mapped_index] = 1; // change dirty bit to 1
						 cache[direct_mapped_index] = data; // write the new data in cache
					}
					else					
					{
						dirty_bit[direct_mapped_index] = 0; // if data is already there then dirty bit is 0 since  no change was done
					}
					valid_bit[direct_mapped_index] = 1; // now that we have valid data, the valid bit is 1
					
					cache_write_hit++;
			   }
			 else // its a write miss
			   {
			 

					 cache[direct_mapped_index] = data; // write new data to cache 
					 
					 cache_tag[direct_mapped_index] = direct_mapped_tag; // update the cache with new tag

					 dirty_bit[direct_mapped_index] = 0; // dirty bit is 0 as this is the first time we have the new value from write operation but we put that value in memory just now

					 valid_bit[direct_mapped_index] = 1; // now that valid data is in cache, valid bit becomes 1 
					 
					 cache_write_miss++;
					}

		}
			
		
		two_way_index = (address >> 4)& two_mask;
		two_way_tag = address >> 12;
		
		// 2 way set associative
		
		if(read)
		{	
			found = 0;	
		/*	for(a = 0; a < 2; a++) // going to appropriate set and looping through
			{
				if ((two_way_tag == two_cache_tag[(two_way_index << 1) | a]) && two_valid_bit[(two_way_index << 1) | a]) // the address tag matches cache tag and data is valid, i.e it is what we want
				{
					
					two_cache_read_hit++;// then its a hit
					found = 1; // data was found
					// processor uses that data
					temp_lru = lru_counter[(two_way_index << 1) | a]; // keeping track of the lru value of the location which was operated on
					
					temp_index = (two_way_index << 1) | a; // keeping track of the location which was operated on
					
					break;
				}	
			} */
			
			 
			 hipMalloc((void**)&dev_two_way_tag, sizeof(unsigned));
			 hipMalloc((void**)&dev_two_cache_tag, 512*sizeof(unsigned));
			 hipMalloc((void**)&dev_two_way_index, sizeof(unsigned));
			 hipMalloc((void**)&dev_two_valid_bit, 512*sizeof(int));
			 hipMalloc((void**)&dev_two_cache_read_hit, sizeof(int));
			 hipMalloc((void**)&dev_found, sizeof(int));
			 hipMalloc((void**)&dev_temp_lru, sizeof(int));
			 hipMalloc((void**)&dev_lru_counter, 512*sizeof(int));
			 hipMalloc((void**)&dev_temp_index, sizeof(int));
			
			 hipMemcpy(dev_two_way_tag, two_way_tag, sizeof(unsigned), hipMemcpyHostToDevice);
			 hipMemcpy(dev_two_cache_tag, two_cache_tag, 512*sizeof(unsigned), hipMemcpyHostToDevice);
			 hipMemcpy(dev_two_way_index, two_way_index, sizeof(unsigned), hipMemcpyHostToDevice);
			 hipMemcpy(dev_two_valid_bit, two_valid_bit, 512*sizeof(int), hipMemcpyHostToDevice);
			 hipMemcpy(dev_two_cache_read_hit, two_cache_read_hit, sizeof(int), hipMemcpyHostToDevice);
			 hipMemcpy(dev_found, found, sizeof(int), hipMemcpyHostToDevice);
			 hipMemcpy(dev_temp_lru, temp_lru, sizeof(int), hipMemcpyHostToDevice);
			 hipMemcpy(dev_lru_counter, lru_counter, 512*sizeof(int), hipMemcpyHostToDevice);
			 hipMemcpy(dev_temp_index, temp_index, sizeof(int), hipMemcpyHostToDevice);
						
			read_hit<<<1,2>>>(dev_two_way_tag, dev_two_cache_tag, dev_two_way_index, dev_two_valid_bit, dev_two_cache_read_hit, dev_found, dev_temp_lru, dev_lru_counter, dev_temp_index);
			
			 hipMemcpy(two_way_tag, dev_two_way_tag, sizeof(unsigned), hipMemcpyDeviceToHost);
			 hipMemcpy(two_cache_tag, dev_two_cache_tag, 512*sizeof(unsigned), hipMemcpyDeviceToHost);
			 hipMemcpy(two_way_index, dev_two_way_index, sizeof(unsigned), hipMemcpyDeviceToHost);
			 hipMemcpy(two_valid_bit, dev_two_valid_bit, 512*sizeof(int), hipMemcpyDeviceToHost);
			 hipMemcpy(two_cache_read_hit, dev_two_cache_read_hit,  sizeof(int), hipMemcpyDeviceToHost);
			 hipMemcpy(found, dev_found, sizeof(int), hipMemcpyDeviceToHost);
			 hipMemcpy(temp_lru, dev_temp_lru, sizeof(int), hipMemcpyDeviceToHost);
			 hipMemcpy(lru_counter, dev_lru_counter, 512*sizeof(int), hipMemcpyDeviceToHost);
			 hipMemcpy(temp_index, dev_temp_index, sizeof(int), hipMemcpyDeviceToHost);
			 
			 hipFree(dev_two_way_tag);
			 hipFree(dev_two_cache_tag);
			 hipFree(dev_two_way_index);
			 hipFree(dev_two_valid_bit);
			 hipFree(dev_two_cache_read_hit);
			 hipFree(dev_found);
			 hipFree(dev_temp_lru);
			 hipFree(dev_lru_counter);
			 hipFree(dev_temp_index);
			
			if(!found)  // if its a miss
			{
				two_cache_read_miss++;
			/*	for(a=0;a<2;a++)  // we see if any locations in the set were empty
				{
					if(two_cache[(two_way_index << 1) | a] == 0 && lru_counter[(two_way_index << 1) | a] == h_lru_count[two_way_index]) // place it in empty spot with highest LRU count for empty spot							
						{
							two_cache[(two_way_index << 1) | a] = data; // get the value from memory and store it in location with highest lru value for empty spot
							
							two_cache_tag[(two_way_index << 1) | a] = two_way_tag; // update the tag in cache
							
							two_valid_bit[(two_way_index << 1) | a] = 1; // set the valid bit to 1 as its the first time value is loaded in cache
							
							two_dirty_bit[(two_way_index << 1) | a] = 0; // setting dirty bit to 0 as new value was loaded into cache from memory
							
							h_lru_count[two_way_index]--; // the highest lru count for an empty spot is now decreased by 1
							
							temp_lru = lru_counter[(two_way_index << 1) | a]; // keeping track of the lru value of the location which was operated on
							
							temp_index = (two_way_index << 1) | a; // keeping track of the location which was operated on
							
							break; // exiting the loop when our goal was achieved
						}	
				}  */
				
				 hipMalloc((void**)&dev_two_cache, 512*sizeof(int));
				 hipMalloc((void**)&dev_two_way_index, sizeof(unsigned));
				 hipMalloc((void**)&dev_two_cache_tag, 512*sizeof(unsigned));
				 hipMalloc((void**)&dev_two_valid_bit, 512*sizeof(int));
				 hipMalloc((void**)&dev_two_dirty_bit, 512*sizeof(int));
				 hipMalloc((void**)&dev_lru_counter, 512*sizeof(int));
				 hipMalloc((void**)&dev_h_lru_count, 256*sizeof(int));
				 hipMalloc((void**)&dev_two_way_tag, sizeof(unsigned));
				 hipMalloc((void**)&dev_data, sizeof(int));
				 hipMalloc((void**)&dev_temp_lru, sizeof(int));
				 hipMalloc((void**)&dev_temp_index, sizeof(int));
				 
	
				 hipMemcpy(dev_two_cache, two_cache, 512*sizeof(int), hipMemcpyHostToDevice);				
				 hipMemcpy(dev_two_cache_tag, two_cache_tag, 512*sizeof(unsigned), hipMemcpyHostToDevice);			
				 hipMemcpy(dev_two_valid_bit, two_valid_bit, 512*sizeof(int), hipMemcpyHostToDevice);				
				 hipMemcpy(dev_two_dirty_bit, two_dirty_bit, 512*sizeof(int), hipMemcpyHostToDevice);
				 hipMemcpy(dev_lru_counter, lru_counter, 512*sizeof(int), hipMemcpyHostToDevice);
				 hipMemcpy(dev_h_lru_count, h_lru_count, 256*sizeof(int), hipMemcpyHostToDevice);
				 hipMemcpy(dev_two_way_tag, two_way_tag, sizeof(unsigned), hipMemcpyHostToDevice);				
				 hipMemcpy(dev_two_way_index, two_way_index, sizeof(unsigned), hipMemcpyHostToDevice);				 
				 hipMemcpy(dev_data, data, sizeof(int), hipMemcpyHostToDevice);			 
				 hipMemcpy(dev_temp_lru, temp_lru, sizeof(int), hipMemcpyHostToDevice);
				 hipMemcpy(dev_temp_index, temp_index, sizeof(int), hipMemcpyHostToDevice);
					 				 
												
				read_miss_empty<<<1,2>>>(dev_two_cache, dev_two_way_index, dev_two_cache_tag, dev_two_valid_bit, dev_two_dirty_bit, dev_lru_counter, dev_h_lru_count, dev_temp_lru, dev_temp_index, dev_data, two_way_tag);
				
				 
				 hipMemcpy(two_cache, dev_two_cache, 512*sizeof(int), hipMemcpyDeviceToHost);
				 hipMemcpy(two_cache_tag, dev_two_cache_tag, 512*sizeof(unsigned), hipMemcpyDeviceToHost);
				 hipMemcpy(two_valid_bit, dev_two_valid_bit, 512*sizeof(int), hipMemcpyDeviceToHost);
				 hipMemcpy(two_dirty_bit, dev_two_dirty_bit, 512*sizeof(int), hipMemcpyDeviceToHost);
				 hipMemcpy(lru_counter, dev_lru_counter, 512*sizeof(int), hipMemcpyDeviceToHost);
				 hipMemcpy(h_lru_count, dev_h_lru_count, 256*sizeof(int), hipMemcpyDeviceToHost);				
				 hipMemcpy(two_way_tag, dev_two_way_tag, sizeof(unsigned), hipMemcpyDeviceToHost);
				 hipMemcpy(data, dev_data, sizeof(int), hipMemcpyDeviceToHost);
				 hipMemcpy(two_way_index, dev_two_way_index, sizeof(unsigned), hipMemcpyDeviceToHost);
				 hipMemcpy(temp_lru, dev_temp_lru, sizeof(int), hipMemcpyDeviceToHost);
				 hipMemcpy(temp_index, dev_temp_index, sizeof(int), hipMemcpyDeviceToHost);
					
				hipFree(dev_two_cache);
				hipFree(dev_two_way_index);
				hipFree(dev_two_cache_tag);
				hipFree(dev_two_valid_bit);
				hipFree(dev_two_dirty_bit);
				hipFree(dev_lru_counter);
				hipFree(dev_h_lru_count);
				hipFree(dev_temp_lru);
				hipFree(dev_temp_index);
				hipFree(dev_data);
				hipFree(dev_two_way_tag);
				
				if(h_lru_count[two_way_index] == 0) // when set is no longer empty
				{
					
				/*	for(a=0;a<2;a++) // we want to see which location in set has to be replaced
					{
						if (lru_counter[(two_way_index << 1) | a]==1) // check for location which was least recently used
						{
							
							two_cache[(two_way_index << 1) | a] = data; // replace the location which was least recently used with data from memory
							
							two_cache_tag[(two_way_index << 1) | a] = two_way_tag; // update the tag in cache
							
							temp_lru = lru_counter[(two_way_index << 1) | a]; // keeping track of the lru value of the location which was operated on
							
							temp_index = (two_way_index << 1) | a; // keeping track of the location which was operated on
							
							two_dirty_bit[(two_way_index << 1) | a] = 0; // setting dirty bit to 0 as new value was loaded into cache from memory
							
							two_valid_bit[(two_way_index << 1) | a] = 1; // data in cache is valid
							
							break; // exiting the loop when our goal was achieved
						}
					} */
					
					
				 hipMalloc((void**)&dev_two_cache, 512*sizeof(int));
				 hipMalloc((void**)&dev_two_way_index, sizeof(unsigned));
				 hipMalloc((void**)&dev_two_cache_tag, 512*sizeof(unsigned));
				 hipMalloc((void**)&dev_two_valid_bit, 512*sizeof(int));
				 hipMalloc((void**)&dev_two_dirty_bit, 512*sizeof(int));
				 hipMalloc((void**)&dev_lru_counter, 512*sizeof(int));
				 
				 hipMalloc((void**)&dev_two_way_tag, sizeof(unsigned));
				 hipMalloc((void**)&dev_data, sizeof(int));
				 hipMalloc((void**)&dev_temp_lru, sizeof(int));
				 hipMalloc((void**)&dev_temp_index, sizeof(int));
				 
	
				 hipMemcpy(dev_two_cache, two_cache, 512*sizeof(int), hipMemcpyHostToDevice);
				 hipMemcpy(dev_two_cache_tag, two_cache_tag, 512*sizeof(unsigned), hipMemcpyHostToDevice);			
				 hipMemcpy(dev_two_valid_bit, two_valid_bit, 512*sizeof(int), hipMemcpyHostToDevice);				
				 hipMemcpy(dev_two_dirty_bit, two_dirty_bit, 512*sizeof(int), hipMemcpyHostToDevice);
				 hipMemcpy(dev_lru_counter, lru_counter, 512*sizeof(int), hipMemcpyHostToDevice);				
				 hipMemcpy(dev_two_way_tag, two_way_tag, sizeof(unsigned), hipMemcpyHostToDevice);				
				 hipMemcpy(dev_two_way_index, two_way_index, sizeof(unsigned), hipMemcpyHostToDevice);				 
				 hipMemcpy(dev_data, data, sizeof(int), hipMemcpyHostToDevice);				 
				 hipMemcpy(dev_temp_lru, temp_lru, sizeof(int), hipMemcpyHostToDevice);				 
				 hipMemcpy(dev_temp_index, temp_index, sizeof(int), hipMemcpyHostToDevice);
					 
					
					read_miss_full<<<1,2>>>(dev_two_cache, dev_two_way_index, dev_two_cache_tag, dev_two_valid_bit, dev_two_dirty_bit, dev_lru_counter, dev_temp_lru, dev_temp_index, dev_data, two_way_tag);
					
					 hipMemcpy(two_cache, dev_two_cache, 512*sizeof(int), hipMemcpyDeviceToHost);
				 hipMemcpy(two_cache_tag, dev_two_cache_tag, 512*sizeof(unsigned), hipMemcpyDeviceToHost);
				 hipMemcpy(two_valid_bit, dev_two_valid_bit, 512*sizeof(int), hipMemcpyDeviceToHost);
				 hipMemcpy(two_dirty_bit, dev_two_dirty_bit, 512*sizeof(int), hipMemcpyDeviceToHost);
				 hipMemcpy(lru_counter, dev_lru_counter, 512*sizeof(int), hipMemcpyDeviceToHost);
				 			
				 hipMemcpy(two_way_tag, dev_two_way_tag, sizeof(unsigned), hipMemcpyDeviceToHost);
				 hipMemcpy(data, dev_data, sizeof(int), hipMemcpyDeviceToHost);
				 hipMemcpy(two_way_index, dev_two_way_index, sizeof(unsigned), hipMemcpyDeviceToHost);
				 hipMemcpy(temp_lru, dev_temp_lru, sizeof(int), hipMemcpyDeviceToHost);
				 hipMemcpy(temp_index, dev_temp_index, sizeof(int), hipMemcpyDeviceToHost);
					
				hipFree(dev_two_cache);
				hipFree(dev_two_way_index);
				hipFree(dev_two_cache_tag);
				hipFree(dev_two_valid_bit);
				hipFree(dev_two_dirty_bit);
				hipFree(dev_lru_counter);
				hipFree(dev_temp_lru);
				hipFree(dev_temp_index);
				hipFree(dev_data);
				hipFree(dev_two_way_tag);
					
				}
				
			}	
			
		}
		else // write statements go over here  
		 //we're implementing write-back and write allocate
		{
			
		  w_found=0;
		  
		/*  for(a=0;a<2;a++) // looping through the set 
		  {
			  if((two_cache_tag[(two_way_index << 1) | a] == two_way_tag) && (two_valid_bit[(two_way_index << 1) | a] == 1))  // seeing if the location we wanna write to (we know this from tag) is already present 
			  {
				  w_found = 1; // we found it
				  two_cache_write_hit++;
				    
				  two_valid_bit[(two_way_index << 1) | a] = 1; 
				  
				  if(two_cache[(two_way_index << 1) | a] != data) // if the data sent by processor is new, i.e not same as one found for that tag
				  {
					  two_cache[(two_way_index << 1) | a] = data; // write new data in cache
					  two_dirty_bit[(two_way_index << 1) | a] = 1; // set dirty bit to one as new data was found
				  }
				  else
				  { 
					two_dirty_bit[(two_way_index << 1) | a] = 0; // if processor didn't provide new data, set dirty bit to 0
				  }
				  
				temp_lru = lru_counter[(two_way_index << 1) | a]; // keeping track of the lru value of the location which was operated on
				
				temp_index = (two_way_index << 1) | a; // keeping track of the location which was operated on						

				break;	
			  }
		  }  */
		  
		  hipMalloc((void**)&dev_two_way_tag, sizeof(unsigned));
			 hipMalloc((void**)&dev_two_cache_tag, 512*sizeof(unsigned));
			 hipMalloc((void**)&dev_two_way_index, sizeof(unsigned));
			 hipMalloc((void**)&dev_two_valid_bit, 512*sizeof(int));
			 hipMalloc((void**)&dev_two_cache_read_hit, sizeof(int));
			 hipMalloc((void**)&dev_w_found, sizeof(int));
			 hipMalloc((void**)&dev_temp_lru, sizeof(int));
			 hipMalloc((void**)&dev_lru_counter, 512*sizeof(int));
			 hipMalloc((void**)&dev_temp_index, sizeof(int));
			 hipMalloc((void**)&dev_data, sizeof(int));
			
			 hipMemcpy(dev_two_way_tag, two_way_tag, sizeof(unsigned), hipMemcpyHostToDevice);
			 hipMemcpy(dev_two_cache_tag, two_cache_tag, 512*sizeof(unsigned), hipMemcpyHostToDevice);
			 hipMemcpy(dev_two_way_index, two_way_index, sizeof(unsigned), hipMemcpyHostToDevice);
			 hipMemcpy(dev_two_valid_bit, two_valid_bit, 512*sizeof(int), hipMemcpyHostToDevice);
			 hipMemcpy(dev_two_cache_read_hit, two_cache_read_hit, sizeof(int), hipMemcpyHostToDevice);
			 hipMemcpy(dev_w_found, w_found, sizeof(int), hipMemcpyHostToDevice);
			 hipMemcpy(dev_temp_lru, temp_lru, sizeof(int), hipMemcpyHostToDevice);
			 hipMemcpy(dev_lru_counter, lru_counter, 512*sizeof(int), hipMemcpyHostToDevice);
			 hipMemcpy(dev_temp_index, temp_index, sizeof(int), hipMemcpyHostToDevice);
			 hipMemcpy(dev_data, data, sizeof(int), hipMemcpyHostToDevice);
		  
		  write_hit<<<1,2>>>(dev_two_way_tag, dev_two_cache_tag, dev_two_way_index, dev_two_valid_bit, dev_two_cache_write_hit, dev_w_found, dev_temp_lru, dev_lru_counter, dev_temp_index, dev_data);
		  
		   hipMemcpy(two_way_tag, dev_two_way_tag, sizeof(unsigned), hipMemcpyDeviceToHost);
			 hipMemcpy(two_cache_tag, dev_two_cache_tag, 512*sizeof(unsigned), hipMemcpyDeviceToHost);
			 hipMemcpy(two_way_index, dev_two_way_index, sizeof(unsigned), hipMemcpyDeviceToHost);
			 hipMemcpy(two_valid_bit, dev_two_valid_bit, 512*sizeof(int), hipMemcpyDeviceToHost);
			 hipMemcpy(two_cache_read_hit, dev_two_cache_read_hit,  sizeof(int), hipMemcpyDeviceToHost);
			 hipMemcpy(w_found, dev_w_found, sizeof(int), hipMemcpyDeviceToHost);
			 hipMemcpy(temp_lru, dev_temp_lru, sizeof(int), hipMemcpyDeviceToHost);
			 hipMemcpy(lru_counter, dev_lru_counter, 512*sizeof(int), hipMemcpyDeviceToHost);
			 hipMemcpy(temp_index, dev_temp_index, sizeof(int), hipMemcpyDeviceToHost);
			 hipMemcpy(data, dev_data, sizeof(int), hipMemcpyDeviceToHost);
			 
			 hipFree(dev_two_way_tag);
			 hipFree(dev_two_cache_tag);
			 hipFree(dev_two_way_index);
			 hipFree(dev_two_valid_bit);
			 hipFree(dev_two_cache_read_hit);
			 hipFree(dev_w_found);
			 hipFree(dev_temp_lru);
			 hipFree(dev_lru_counter);
			 hipFree(dev_temp_index);
			 hipFree(dev_data);
			
		  
		  if(!w_found) // if that tag wasn't found, 
			  
			  {
				  two_cache_write_miss++;
				
				/*  for(a=0;a<2;a++) // loop through the set 
				  {
				  
					if(two_cache[(two_way_index << 1) | a]==0 && lru_counter[(two_way_index << 1) | a] == h_lru_count[two_way_index]) // we search for an empty location with highest LRU count for an empty spot
					{
						two_cache[(two_way_index << 1) | a] = data; // load the new data given by processor in cache
						
						
						two_cache_tag[(two_way_index << 1) | a] = two_way_tag; // update the tag in cache
						
						two_valid_bit[(two_way_index << 1) | a] = 1; // set the valid bit to 1 as its the first time value is loaded in cache
						
						two_dirty_bit[(two_way_index << 1) | a] = 0; // setting dirty bit to 0 as value loaded in cache was just loaded in memory too
						
						h_lru_count[two_way_index]--; // the highest lru count for an empty spot is now decreased by 1
						
						temp_lru = lru_counter[(two_way_index << 1) | a]; // keeping track of the lru value of the location which was operated on
						
						temp_index = (two_way_index << 1) | a; // keeping track of the location which was operated on
						
						break; // exiting the loop when our goal was achieved
					}
				  } */
				  
				  
				   hipMalloc((void**)&dev_two_cache, 512*sizeof(int));
				 hipMalloc((void**)&dev_two_way_index, sizeof(unsigned));
				 hipMalloc((void**)&dev_two_cache_tag, 512*sizeof(unsigned));
				 hipMalloc((void**)&dev_two_valid_bit, 512*sizeof(int));
				 hipMalloc((void**)&dev_two_dirty_bit, 512*sizeof(int));
				 hipMalloc((void**)&dev_lru_counter, 512*sizeof(int));
				 hipMalloc((void**)&dev_h_lru_count, 256*sizeof(int));
				 hipMalloc((void**)&dev_two_way_tag, sizeof(unsigned));
				 hipMalloc((void**)&dev_data, sizeof(int));
				 hipMalloc((void**)&dev_temp_lru, sizeof(int));
				 hipMalloc((void**)&dev_temp_index, sizeof(int));
				 
	
				 hipMemcpy(dev_two_cache, two_cache, 512*sizeof(int), hipMemcpyHostToDevice);				
				 hipMemcpy(dev_two_cache_tag, two_cache_tag, 512*sizeof(unsigned), hipMemcpyHostToDevice);			
				 hipMemcpy(dev_two_valid_bit, two_valid_bit, 512*sizeof(int), hipMemcpyHostToDevice);				
				 hipMemcpy(dev_two_dirty_bit, two_dirty_bit, 512*sizeof(int), hipMemcpyHostToDevice);
				 hipMemcpy(dev_lru_counter, lru_counter, 512*sizeof(int), hipMemcpyHostToDevice);
				 hipMemcpy(dev_h_lru_count, h_lru_count, 256*sizeof(int), hipMemcpyHostToDevice);
				 hipMemcpy(dev_two_way_tag, two_way_tag, sizeof(unsigned), hipMemcpyHostToDevice);				
				 hipMemcpy(dev_two_way_index, two_way_index, sizeof(unsigned), hipMemcpyHostToDevice);				 
				 hipMemcpy(dev_data, data, sizeof(int), hipMemcpyHostToDevice);				 
				 hipMemcpy(dev_temp_lru, temp_lru, sizeof(int), hipMemcpyHostToDevice);				 
				 hipMemcpy(dev_temp_index, temp_index, sizeof(int), hipMemcpyHostToDevice);
					 
				  
				  write_miss_empty<<<1,2>>>(dev_two_cache, dev_two_way_index, dev_two_cache_tag, dev_two_valid_bit, dev_two_dirty_bit, dev_lru_counter, dev_h_lru_count, dev_temp_lru, dev_temp_index, dev_data, two_way_tag);
				  
				  
				 hipMemcpy(two_cache, dev_two_cache, 512*sizeof(int), hipMemcpyDeviceToHost);
				 hipMemcpy(two_cache_tag, dev_two_cache_tag, 512*sizeof(unsigned), hipMemcpyDeviceToHost);
				 hipMemcpy(two_valid_bit, dev_two_valid_bit, 512*sizeof(int), hipMemcpyDeviceToHost);
				 hipMemcpy(two_dirty_bit, dev_two_dirty_bit, 512*sizeof(int), hipMemcpyDeviceToHost);
				 hipMemcpy(lru_counter, dev_lru_counter, 512*sizeof(int), hipMemcpyDeviceToHost);
				 hipMemcpy(h_lru_count, dev_h_lru_count, 256*sizeof(int), hipMemcpyDeviceToHost);				
				 hipMemcpy(two_way_tag, dev_two_way_tag, sizeof(unsigned), hipMemcpyDeviceToHost);
				 hipMemcpy(data, dev_data, sizeof(int), hipMemcpyDeviceToHost);
				 hipMemcpy(two_way_index, dev_two_way_index, sizeof(unsigned), hipMemcpyDeviceToHost);
				 hipMemcpy(temp_lru, dev_temp_lru, sizeof(int), hipMemcpyDeviceToHost);
				 hipMemcpy(temp_index, dev_temp_index, sizeof(int), hipMemcpyDeviceToHost);
					
				hipFree(dev_two_cache);
				hipFree(dev_two_way_index);
				hipFree(dev_two_cache_tag);
				hipFree(dev_two_valid_bit);
				hipFree(dev_two_dirty_bit);
				hipFree(dev_lru_counter);
				hipFree(dev_h_lru_count);
				hipFree(dev_temp_lru);
				hipFree(dev_temp_index);
				hipFree(dev_data);
				hipFree(dev_two_way_tag);
				
				if(h_lru_count[two_way_index] == 0) //all the blocks are occupied
					{
							
					/*	for(a=0;a<2;a++) // we want to see which location in set has to be replaced
						{
							if (lru_counter[(two_way_index << 1) | a] == 1) // check for location which was least recently used
							{
								
								two_cache[(two_way_index << 1) | a] = data; // replace the location which was least recently used with data from processor
								
								two_cache_tag[(two_way_index << 1) | a] = two_way_tag; // update the tag in cache
								
								temp_lru = lru_counter[(two_way_index << 1) | a]; // keeping track of the lru value of the location which was operated on
								
								temp_index = (two_way_index << 1) | a; // keeping track of the location which was operated on
								
								two_dirty_bit[(two_way_index << 1) | a] = 0; // setting dirty bit to 0 as the value loaded in cache was also just loaded in memory
								
								two_valid_bit[(two_way_index << 1) | a] = 1; // the data is valid
								
								break; // exiting the loop when our goal was achieved
							}
						}  */
						
						
						 hipMalloc((void**)&dev_two_cache, 512*sizeof(int));
				 hipMalloc((void**)&dev_two_way_index, sizeof(unsigned));
				 hipMalloc((void**)&dev_two_cache_tag, 512*sizeof(unsigned));
				 hipMalloc((void**)&dev_two_valid_bit, 512*sizeof(int));
				 hipMalloc((void**)&dev_two_dirty_bit, 512*sizeof(int));
				 hipMalloc((void**)&dev_lru_counter, 512*sizeof(int));
				 
				 hipMalloc((void**)&dev_two_way_tag, sizeof(unsigned));
				 hipMalloc((void**)&dev_data, sizeof(int));
				 hipMalloc((void**)&dev_temp_lru, sizeof(int));
				 hipMalloc((void**)&dev_temp_index, sizeof(int));
				 
	
				 hipMemcpy(dev_two_cache, two_cache, 512*sizeof(int), hipMemcpyHostToDevice);			
				 hipMemcpy(dev_two_cache_tag, two_cache_tag, 512*sizeof(unsigned), hipMemcpyHostToDevice);			
				 hipMemcpy(dev_two_valid_bit, two_valid_bit, 512*sizeof(int), hipMemcpyHostToDevice);				
				 hipMemcpy(dev_two_dirty_bit, two_dirty_bit, 512*sizeof(int), hipMemcpyHostToDevice);
				 hipMemcpy(dev_lru_counter, lru_counter, 512*sizeof(int), hipMemcpyHostToDevice);				
				 hipMemcpy(dev_two_way_tag, two_way_tag, sizeof(unsigned), hipMemcpyHostToDevice);			
				 hipMemcpy(dev_two_way_index, two_way_index, sizeof(unsigned), hipMemcpyHostToDevice);			 
				 hipMemcpy(dev_data, data, sizeof(int), hipMemcpyHostToDevice);				 
				 hipMemcpy(dev_temp_lru, temp_lru, sizeof(int), hipMemcpyHostToDevice);				 
				 hipMemcpy(dev_temp_index, temp_index, sizeof(int), hipMemcpyHostToDevice);
					 
					
						
						write_miss_full<<<1,2>>>(dev_two_cache, dev_two_way_index, dev_two_cache_tag, dev_two_valid_bit, dev_two_dirty_bit, dev_lru_counter, dev_temp_lru, dev_temp_index, dev_data, two_way_tag);
						
						 hipMemcpy(two_cache, dev_two_cache, 512*sizeof(int), hipMemcpyDeviceToHost);
				 hipMemcpy(two_cache_tag, dev_two_cache_tag, 512*sizeof(unsigned), hipMemcpyDeviceToHost);
				 hipMemcpy(two_valid_bit, dev_two_valid_bit, 512*sizeof(int), hipMemcpyDeviceToHost);
				 hipMemcpy(two_dirty_bit, dev_two_dirty_bit, 512*sizeof(int), hipMemcpyDeviceToHost);
				 hipMemcpy(lru_counter, dev_lru_counter, 512*sizeof(int), hipMemcpyDeviceToHost);
				 			
				 hipMemcpy(two_way_tag, dev_two_way_tag, sizeof(unsigned), hipMemcpyDeviceToHost);
				 hipMemcpy(data, dev_data, sizeof(int), hipMemcpyDeviceToHost);
				 hipMemcpy(two_way_index, dev_two_way_index, sizeof(unsigned), hipMemcpyDeviceToHost);
				 hipMemcpy(temp_lru, dev_temp_lru, sizeof(int), hipMemcpyDeviceToHost);
				 hipMemcpy(temp_index, dev_temp_index, sizeof(int), hipMemcpyDeviceToHost);
					
				hipFree(dev_two_cache);
				hipFree(dev_two_way_index);
				hipFree(dev_two_cache_tag);
				hipFree(dev_two_valid_bit);
				hipFree(dev_two_dirty_bit);
				hipFree(dev_lru_counter);
				hipFree(dev_temp_lru);
				hipFree(dev_temp_index);
				hipFree(dev_data);
				hipFree(dev_two_way_tag);
						
					}
			  }
									
		}
					
		/*	for(a=0;a<2;a++) // looping through the lru counters for the particular set for updating
				
			{
				if(lru_counter[(two_way_index << 1) | a] > temp_lru) // if any lru value is greater than the value of location operated upon
					lru_counter[(two_way_index << 1) | a]--; // decrement it by 1
			} */
			
			hipMalloc((void**)&dev_lru_counter, 512*sizeof(int));
			hipMalloc((void**)&dev_two_way_index, sizeof(int));
			hipMalloc((void**)&dev_temp_lru, sizeof(int));
			
			hipMemcpy(dev_lru_counter, lru_counter, 512*sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(dev_two_way_index, two_way_index, sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(dev_temp_lru, temp_lru, sizeof(int), hipMemcpyHostToDevice);
			
			lru_count<<<1,2>>>(dev_lru_counter, dev_two_way_index, dev_temp_lru);
			
			hipMemcpy(lru_counter, dev_lru_counter, 512*sizeof(int), hipMemcpyDeviceToHost);
			hipMemcpy(two_way_index, dev_two_way_index, sizeof(int), hipMemcpyDeviceToHost);
			hipMemcpy(temp_lru, dev_temp_lru, sizeof(int), hipMemcpyDeviceToHost);
			
			hipFree(dev_lru_counter);
			hipFree(dev_two_way_index);
			hipFree(dev_temp_lru);
			
			lru_counter[temp_index] = 2; // set the lru to be highest for the location which was just operated upon
		
			
			
			
			
   }
      
 printf(" The total cache read hits for direct mapped = %d \n",cache_read_hit); 
 printf(" The total cache read misses for direct mapped are = %d \n",cache_read_miss); 
 printf(" The total cache write hit for direct mapped are = %d \n",cache_write_hit); 
 printf(" The total cache write misses for direct mapped are = %d \n",cache_write_miss); 
 printf("\n");
 printf(" The total cache read hits for two way set associativity = %d \n",two_cache_read_hit); 
 printf(" The total cache read misses for two way set associativity = %d \n",two_cache_read_miss); 
 printf(" The total cache write hit for two way set associativity are = %d \n",two_cache_write_hit); 
 printf(" The total cache write misses for two way set associativity are = %d \n",two_cache_write_miss); 

return 0;
}

